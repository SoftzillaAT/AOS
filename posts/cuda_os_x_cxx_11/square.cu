// Copyright (c) 2013 Craig Wright (kungfucraig@gmail.com)
// 
// Permission is hereby granted, free of charge, to any person obtaining a copy of
// this software and associated documentation files (the "Software"), to deal in
// the Software without restriction, including without limitation the rights to use,
// copy, modify, merge, publish, distribute, sublicense, and/or sell copies of the
// Software, and to permit persons to whom the Software is furnished to do so,
// subject to the following conditions:
// 
// The above copyright notice and this permission notice shall be included in all copies
// or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED,
// INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS FOR A 
// PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT
// HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION
// OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE
// OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.


#include <hip/hip_runtime.h>

//! Square the data.
__global__ void square(float *data)
{
   const unsigned int tid = threadIdx.x;
   float d = data[tid];
   data[tid] = d*d;
}

//! Provide a wrapper for the kernel.
extern "C" hipError_t runKernel(int len, float *dataD)
{
   dim3 grid(1, 1, 1);
   dim3 threads(len, 1, 1);
   square<<< grid, threads >>>(dataD);
   return hipGetLastError();
}


