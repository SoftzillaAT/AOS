
#include <hip/hip_runtime.h>
#include <stdio.h>

//! Square the data.
__global__ void
square(float *data)
{
   const unsigned int tid = threadIdx.x;
   float d = data[tid];
   data[tid] = d*d;
}

//! Provide a wrapper for the kernel.
extern "C"
hipError_t runKernel(int len, float *dataD)
{
   dim3 grid(1, 1, 1);
   dim3 threads(len, 1, 1);
   square<<< grid, threads >>>(dataD);
   return hipGetLastError();
}


